#include "hip/hip_runtime.h"
#include <iostream>
#include "book.h"

#define N 10

__global__ void add(int *a, int *b, int *c) {
  int tid = blockIdx.x;
  if (tid < N) {
    c[tid] = a[tid] + b[tid];
  }
}

__global__ void fill(int *a, int *b) {
  int tid = blockIdx.x;
  printf("tid: %d\n", tid); 
  if (tid < N) {
    a[tid] = -tid;
    b[tid] = tid * tid;
  }
}


int main() {
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;

  // Allocate device memory.
  hipMalloc((void **)&dev_a, N * sizeof(int));
  hipMalloc((void **)&dev_b, N * sizeof(int));
  hipMalloc((void **)&dev_c, N * sizeof(int));

  // Fill the array "a" and "b" on CPU.
  //for (int i = 0; i < N; ++ i) {
  //  a[i] = -i;
  //  b[i] = i * i;
  //}

  //hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
  //hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

  // Fill the array "a" and "b" on GPU.
  fill<<<N, 1>>>(dev_a, dev_b);
  hipMemcpy(a, dev_a, N * sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(b, dev_b, N * sizeof(int), hipMemcpyDeviceToHost);

  add<<<N, 1>>> (dev_a, dev_b, dev_c);

  // Copy "c" back to CPU.
  hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

  // Display the results.
  for (int i = 0; i < N; ++ i) {
    std::cout << a[i] << " + " << b[i] << " = " << c[i] << std::endl;
  }
  return 0;
}

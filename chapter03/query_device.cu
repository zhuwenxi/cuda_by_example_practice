#include <iostream>
#include "book.h"

int main() {
  int count;
  HANDLE_ERROR(hipGetDeviceCount(&count));
  
  hipDeviceProp_t prop;
  for (int i = 0; i < count; ++ i) {
    HANDLE_ERROR(hipGetDeviceProperties(&prop, i));
    std::cout << "--- Genernal information for device " << i << " ---" << std::endl;
    std::cout << "Name: " << prop.name << std::endl;
    std::cout << "Compute capability: " << prop.major << "." << prop.minor << std::endl;
    std::cout << "Clock rate: " << prop.clockRate << std::endl;
    std::cout << "Device copy overlap:" << prop.deviceOverlap << std::endl;
    std::cout << "Kernel execution timeout: " << prop.kernelExecTimeoutEnabled << std::endl;

    std::cout << "--- Memory information for device " << i << " ---" << std::endl;
    std::cout << "Total global mem: " << prop.totalGlobalMem << std::endl;
    std::cout << "Total constant mem: " << prop.totalConstMem << std::endl;
    std::cout << "Max mem pitch: " << prop.memPitch << std::endl;
    std::cout << "Texture Alignment: " << prop.textureAlignment << std::endl;

    std::cout << "--- MP Information for device " << i << " ---" << std::endl;
    std::cout << "Multiprocessor count: " << prop.multiProcessorCount << std::endl;
    std::cout << "Shared mem per mp: " << prop.sharedMemPerBlock << std::endl;
    std::cout << "Registers per mp: " << prop.regsPerBlock << std::endl;
    std::cout << "Threads in warp: " << prop.warpSize << std::endl;
    std::cout << "Max threads per block: " << prop.maxThreadsPerBlock << std::endl;
    std::cout << "Max thread dimensions: (" << prop.maxThreadsDim[0] << ", " << prop.maxThreadsDim[1] << ", " << prop.maxThreadsDim[2] << ")" << std::endl;
    std::cout << "Max grid dimensions: (" << prop.maxGridSize[0] << ", " << prop.maxGridSize[1] << ", " << prop.maxGridSize[2] << ")" << std::endl;
    break;
  }
}

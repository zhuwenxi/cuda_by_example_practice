#include <iostream>

#include "book.h"


int main(void) {
  hipDeviceProp_t prop;
  int dev;
  

  hipGetDevice(&dev);
  std::cout << "ID of current CUDA device: " << dev << std::endl;

  memset(&prop, 0, sizeof(hipDeviceProp_t));
  prop.major = 1;
  prop.minor = 3;
  hipChooseDevice(&dev, &prop);
  std::cout << "ID of CUDA device closest to revision 1.3: " << dev << std::endl;
  
  return 0;
}

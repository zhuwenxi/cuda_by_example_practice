#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include "book.h"


#define N (33 * 1024)

__global__ void add(int *a, int *b, int *c) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  while (tid < N) {
    c[tid] = a[tid] + b[tid];
    tid += blockDim.x * gridDim.x;
  }
}

int main() {
  int a[N], b[N], c[N];
  int *dev_a, *dev_b, *dev_c;
  
  hipMalloc((void **)&dev_a, N * sizeof(int));
  hipMalloc((void **)&dev_b, N * sizeof(int));
  hipMalloc((void **)&dev_c, N * sizeof(int));

  for (int i = 0; i < N; i ++) {
    a[i] = i;
    b[i] = i * i;
  }

  hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

  add<<<128, 128>>>(dev_a, dev_b, dev_c);

  hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);

  // Verify that the GPU did the work we requested.
  bool success = true;
  for (int i = 0; i < N; ++ i) {
    if ((a[i] + b[i]) != c[i]) {
      printf("Error: %d + %d != %d\n", a[i], b[i], c[i]);
      success = false;
      break;
    }
  }

  if (success) {
    printf("We did it!\n");
  }

  hipFree(dev_a);
  hipFree(dev_b);
  hipFree(dev_c);

  return 0;
}
